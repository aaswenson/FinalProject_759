#include "hip/hip_runtime.h"
// Allocate memory for ptrac data
particleTrack AllocatePtracData(const particleTrack hdata){
	hipError_t error;
	particleTrack data = hdata;
	int size = hdata.Ntracks * sizeof(float);
	error = hipMalloc((void**)&data.x_pos, size);
	error = hipMalloc((void**)&data.y_pos, size);
	error = hipMalloc((void**)&data.z_pos, size);
	error = hipMalloc((void**)&data.u, size);
	error = hipMalloc((void**)&data.v, size);
	error = hipMalloc((void**)&data.w, size);
	error = hipMalloc((void**)&data.track_length, size);
	if (error != hipSuccess)
	{
		printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	return data;
}

// Allocate memory for mesh data
twoDmesh AllocateMeshData(const twoDmesh hmesh){
	hipError_t error;
	twoDmesh dmesh = hmesh;
	int sizeI = hmesh.NI * sizeof(float);
	int sizeJ = hmesh.NJ * sizeof(float);
	int sizeK = hmesh.NK * sizeof(float);
	int sizeflux = hmesh.NI*hmesh.NJ*hmesh.NK*sizeof(float);

    error = hipMalloc((void**)&dmesh.x, sizeI);
	error = hipMalloc((void**)&dmesh.y, sizeJ);
	error = hipMalloc((void**)&dmesh.z, sizeK);
	error = hipMalloc((void**)&dmesh.flux, sizeflux);
	
    if (error != hipSuccess)
	{
		printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	return dmesh;
}

// copy the ptrac and mesh data to the device
void CopyDatatoDevice(particleTrack data, const particleTrack hdata,
                      twoDmesh dmesh, const twoDmesh hmesh)
{
	int size = hdata.Ntracks * sizeof(float);
	int sizeI = hmesh.NI * sizeof(float);
	int sizeJ = hmesh.NJ * sizeof(float);
	int sizeK = hmesh.NK * sizeof(float);
	int sizeflux = hmesh.NI*hmesh.NJ*hmesh.NK*sizeof(float);
	
    // ptrac data
    data.x_pos = hdata.x_pos;
	data.y_pos = hdata.y_pos;
	data.z_pos = hdata.z_pos;
	data.u = hdata.u;
	data.v = hdata.v;
	data.w = hdata.w;
	data.track_length = hdata.track_length;
    
    // mesh data
    dmesh.x = hmesh.x; dmesh.y = hmesh.y; dmesh.z = hmesh.z;
    dmesh.flux = hmesh.flux;

    // copy all data to device
    // Ptrac Data
	hipMemcpy(data.x_pos, hdata.x_pos, size, 
			hipMemcpyHostToDevice);
	hipMemcpy(data.y_pos, hdata.y_pos, size, 
			hipMemcpyHostToDevice);
	hipMemcpy(data.z_pos, hdata.z_pos, size, 
			hipMemcpyHostToDevice);
	hipMemcpy(data.u, hdata.u, size, 
			hipMemcpyHostToDevice);
	hipMemcpy(data.v, hdata.v, size, 
			hipMemcpyHostToDevice);
	hipMemcpy(data.w, hdata.w, size, 
			hipMemcpyHostToDevice);
	hipMemcpy(data.track_length, hdata.track_length, size, 
			hipMemcpyHostToDevice);
    // Mesh Data
	hipMemcpy(dmesh.x, hmesh.x, sizeI, 
			hipMemcpyHostToDevice);
	hipMemcpy(dmesh.y, hmesh.y, sizeJ, 
			hipMemcpyHostToDevice);
	hipMemcpy(dmesh.z, hmesh.z, sizeK, 
			hipMemcpyHostToDevice);
	hipMemcpy(dmesh.flux, hmesh.flux, sizeflux, 
			hipMemcpyHostToDevice);
}


//compare the data stored in two arrays on the host
bool CompareResults(float* A, float* B, int elements, float eps,float * error)
{
	for(unsigned int i = 0; i < elements; i++){
		float temp = sqrt((A[i]-B[i])*(A[i]-B[i]));
		*error+=temp;
		if(temp>eps){
			return false;
		} 
	}
	return true;
}

