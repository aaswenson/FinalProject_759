#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>

// Allocate memory for ptrac data
particleTrack AllocatePtracData(particleTrack hdata){
	hipError_t error;
	particleTrack data = hdata;
	int size = hdata.Ntracks * sizeof(float);
	error = hipMalloc((void**)&data.x_pos, size);
	error = hipMalloc((void**)&data.y_pos, size);
	error = hipMalloc((void**)&data.z_pos, size);
	error = hipMalloc((void**)&data.u, size);
	error = hipMalloc((void**)&data.v, size);
	error = hipMalloc((void**)&data.w, size);
	error = hipMalloc((void**)&data.track_length, size);
	if (error != hipSuccess)
	{
		printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	return data;
}

// Allocate memory for mesh data
twoDmesh AllocateMeshData(twoDmesh hmesh){
	hipError_t error;
	twoDmesh dmesh = hmesh;
	int sizeI = hmesh.NI * sizeof(float);
	int sizeJ = hmesh.NJ * sizeof(float);
	int sizeK = hmesh.NK * sizeof(float);
	int sizeflux = hmesh.NI*hmesh.NJ*hmesh.NK*sizeof(float);

    error = hipMalloc((void**)&dmesh.x, sizeI);
	hipMalloc((void**)&dmesh.y, sizeJ);
	hipMalloc((void**)&dmesh.z, sizeK);
	hipMalloc((void**)&dmesh.flux, sizeflux);
	
    if (error != hipSuccess)
	{
		printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	return dmesh;
}

// copy the ptrac and mesh data to the device
void CopyDatatoDevice(particleTrack ddata, particleTrack hdata,
                      twoDmesh dmesh, twoDmesh hmesh)
{
    hipError_t error;
	unsigned int size = hdata.Ntracks * sizeof(float);
    unsigned int meshsize = hmesh.NI * sizeof(float);
	unsigned int sizeflux = hmesh.NI*hmesh.NJ*hmesh.NK*sizeof(float);
	

    // copy all data to device
    // Ptrac Data
	error = hipMemcpy(ddata.x_pos, hdata.x_pos, size, 
			hipMemcpyHostToDevice);
	hipMemcpy(ddata.y_pos, hdata.y_pos, size, 
	    hipMemcpyHostToDevice);
	hipMemcpy(ddata.z_pos, hdata.z_pos, size, 
	    hipMemcpyHostToDevice);
	hipMemcpy(ddata.u, hdata.u, size, 
	    hipMemcpyHostToDevice);
	hipMemcpy(ddata.v, hdata.v, size, 
			hipMemcpyHostToDevice);
	hipMemcpy(ddata.w, hdata.w, size, 
			hipMemcpyHostToDevice);
	hipMemcpy(ddata.track_length, hdata.track_length, size, 
			hipMemcpyHostToDevice);
    // Mesh Data
    hipMemcpy(dmesh.x, hmesh.x, meshsize, 
			hipMemcpyHostToDevice);
	hipMemcpy(dmesh.y, hmesh.y, meshsize, 
			hipMemcpyHostToDevice);
	hipMemcpy(dmesh.z, hmesh.z, meshsize, 
			hipMemcpyHostToDevice);
	hipMemcpy(dmesh.flux, hmesh.flux, sizeflux, 
			hipMemcpyHostToDevice);
    if (error != hipSuccess)
	{
		printf("hipMemcpy returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

}


//compare the data stored in two arrays on the host
bool CompareResults(float* A, float* B, int elements, float eps,float * error)
{
	for(unsigned int i = 0; i < elements; i++){
		float temp = sqrt((A[i]-B[i])*(A[i]-B[i]));
		*error+=temp;
		if(temp>eps){
			return false;
		} 
	}
	return true;
}

