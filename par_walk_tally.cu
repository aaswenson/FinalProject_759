#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <cstdio>
#include <cmath>
#include <vector>
#include <fstream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "gen_mesh.hpp"
#include "random_walk.hpp"
#include "read_methods.hpp"
#include "cuda_prep.hpp"
#include "seq_tally.hpp"

    
__global__  void parallel_walk(unsigned int Ngrid, unsigned int N, float* x, float* y, float* z,
                               float h, float* gflux, 
                               float* x_pos, float* y_pos, float* z_pos, 
                               float* u, float* v, float* w, 
                               float* track_length){
    unsigned int x_idx = threadIdx.x; 
    unsigned int y_idx = threadIdx.y;
    unsigned int z_idx = threadIdx.z;;
    unsigned int tl_ID = (x_idx + y_idx*blockDim.x +
            z_idx*blockDim.x*blockDim.y) +
        blockDim.x*blockDim.y*blockDim.z*blockIdx.x;

    float x_0, y_0, z_0, tl;
    float dir_inv[3];
    float tmin, tmax, savet;
    float x_surfs[2], y_surfs[2], z_surfs[2];
    float V = h*h*h; 
    gflux[tl_ID] = 0;



    if(x_idx < Ngrid || y_idx < Ngrid || z_idx < Ngrid){

        // get voxel surfaces
        x_surfs[0] = x[x_idx]; x_surfs[1] = x[x_idx+1];
        y_surfs[0] = y[y_idx]; y_surfs[1] = y[y_idx+1];
        z_surfs[0] = z[z_idx]; z_surfs[1] = z[z_idx+1];

        for (int PID=0; PID<N; PID++){
            // get particle track length
            tl = track_length[PID];
            // inverted direction to be used in ray-box intersection check
            dir_inv[0] =  1 / u[PID];
            dir_inv[1] =  1 / v[PID];
            dir_inv[2] =  1 / w[PID];

            // default assumption is we cross into box
            x_0 = x_pos[PID]; y_0 = y_pos[PID]; z_0 = z_pos[PID];
            
            
            // x goes first
            float txmin = (x_surfs[0] - x_0) * dir_inv[0];
            float txmax = (x_surfs[1] - x_0) * dir_inv[0];
            // if necessary swap within x
            if (txmax < txmin){
                savet = txmax;
                txmax = txmin;
                txmin = savet;
            }
            // distance to cross in y
            float tymin = (y_surfs[0] - y_0) * dir_inv[1];
            float tymax = (y_surfs[1] - y_0) * dir_inv[1];
            // if necessary swap within y
            if (tymax < tymin){
                savet = tymax;
                tymax = tymin;
                tymin = savet;
            }
            // distance to cross in z
            float tzmin = (z_surfs[0] - z_0) * dir_inv[2];
            float tzmax = (z_surfs[1] - z_0) * dir_inv[2];
            // if necessary swap within z
            if (tzmax < tzmin){
                savet = tzmax;
                tzmax = tzmin;
                tzmin = savet;
            }

            // maximum min t is the distance to box entry
            tmin = fmax(txmin, fmax(tymin, tzmin));
            // minimum max t is the distance to box exit
            tmax = fmin(txmax, fmin(tymax, tzmax));

            // select cases only where particle was in voxel
            if ( tmin < tmax && tmax > 0 && tl > tmin){
                // particle through entire voxel
                if (tl > tmax && tmin > 0){gflux[tl_ID] += (tmax - tmin) / V;}
                // particle starts inside voxel, leaves
                if (tmin < 0 && tl > tmax){gflux[tl_ID] += tmax / V;}
                // particle starts outside voxel, end inside
                if (tmax > tl && tmin > 0){gflux[tl_ID] += (tl - tmin) / V;}
                
                // particle starts inside, ends inside
                if (tmax > tl && tmin < 0){gflux[tl_ID] += tl / V;}
            }
        }
    }
}

void par_tally(twoDmesh hmesh, particleTrack hdata, int N, float h){

    particleTrack ddata = AllocatePtracData(hdata);
    twoDmesh dmesh = AllocateMeshData(hmesh);
    CopyDatatoDevice(ddata, hdata, dmesh, hmesh);
    
    // size of flux memory
    int flux_size = N*N*N*sizeof(float);

    int max_dim = 10;
    int grid_dim = (int) N*N*N/1000+1; //N/max_dim + 1;
    if (N < 11){
        grid_dim = 1;
        max_dim = N;
    }
    dim3 dimBlock(max_dim, max_dim, max_dim);
    //dim3 dimGrid(grid_dim, grid_dim, grid_dim);
    dim3 dimGrid(grid_dim, 1, 1);
    
    
    // time the kernel w/out memory transfer
    float elapsedTime_ex; 
    hipEvent_t startEvent_par_ex, stopEvent_par_ex;
	
    hipEventCreate(&startEvent_par_ex);
	hipEventCreate(&stopEvent_par_ex);
    hipEventRecord(startEvent_par_ex,0);

    parallel_walk<<<dimGrid,dimBlock>>> (N, ddata.Ntracks, dmesh.x, dmesh.y, dmesh.z, 
            h, dmesh.flux, ddata.x_pos, ddata.y_pos, ddata.z_pos,
         ddata.u, ddata.v, ddata.w, ddata.track_length);

    // stop timer
	hipEventRecord(stopEvent_par_ex,0);
	hipEventSynchronize(stopEvent_par_ex);   
	hipEventElapsedTime(&elapsedTime_ex, startEvent_par_ex, stopEvent_par_ex);
    
    std::cout << "Parallel w/out mem    "<<elapsedTime_ex << std::endl;
    // copy data back to device
    hipMemcpy(hmesh.flux, dmesh.flux, flux_size, 
			hipMemcpyDeviceToHost);

    // free memory on device and host
    free_dev_mem(dmesh, ddata);
}



int main(int argc, char* argv[]){

    if (argc != 4){
        std::cout << "Usage: N_particles N h" << std::endl;
        return 1;
    }
    const unsigned Np = atoi(argv[1]);
    const unsigned N = atof(argv[2]);
    const float h = atof(argv[3]); 

    if (N%2==0){
        std::cout << "Mesh dimensions must be odd!" << std::endl;
        return 1;
    }
    // generate track histories
    execute_walk(Np);
    // Load particle collision history
    particleTrack hdata = read_array("event_history.txt");
    // generate mesh
    twoDmesh hmesh = gen_mesh(N, h);
    // generate mesh
    twoDmesh seq_mesh = gen_mesh(N, h);
    
    
	//defining variables for timing
	hipEvent_t startEvent_seq, stopEvent_seq, 
                startEvent_par, stopEvent_par;
	hipEventCreate(&startEvent_seq);
	hipEventCreate(&stopEvent_seq);
	hipEventCreate(&startEvent_par);
	hipEventCreate(&stopEvent_par);
	
    float elapsedTime_par, elapsedTime_seq;
    
    // start timer
    hipEventRecord(startEvent_seq,0);
    // sequential tally
    seq_tally(N, hdata, seq_mesh);
    // stop timer
	hipEventRecord(stopEvent_seq,0);
	hipEventSynchronize(stopEvent_seq);   
	hipEventElapsedTime(&elapsedTime_seq, startEvent_seq, stopEvent_seq);

    // start timer
	hipEventRecord(startEvent_par,0);
    // parallel tally
    par_tally(hmesh, hdata, N, h); 
    // stop timer
	hipEventRecord(stopEvent_par,0);
	hipEventSynchronize(stopEvent_par);   
	hipEventElapsedTime(&elapsedTime_par, startEvent_par, stopEvent_par);
    
    // print timing results
    std::cout <<"Parallel w/mem    " << elapsedTime_par << std::endl;
    std::cout <<"Sequential    " << elapsedTime_seq << std::endl;
    

    free(hmesh.flux);
    free(seq_mesh.flux);
    
    return 0;
}
