#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <cstdio>
#include <cmath>
#include <vector>
#include <fstream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <gen_mesh.cpp>
#include <random_walk.cpp>
#include <read_methods.cpp>
#include <cuda_prep.cu>

    
__global__  void parallel_walk(unsigned int Ngrid, unsigned int N, float* x, float* y, float* z,
                               float dx, float dy, float dz, float* gflux, 
                               float* x_pos, float* y_pos, float* z_pos, 
                               float* u, float* v, float* w, 
                               float* track_length){
    unsigned int x_idx = threadIdx.x; 
    unsigned int y_idx = threadIdx.y;
    unsigned int z_idx = threadIdx.z;;
    unsigned int tl_ID = (x_idx + y_idx*blockDim.x +
            z_idx*blockDim.x*blockDim.y) +
        blockDim.x*blockDim.y*blockDim.z*blockIdx.x;

    float x_0, y_0, z_0, tl;
    float dir_inv[3];
    float tmin, tmax, savet;
    float x_surfs[2], y_surfs[2], z_surfs[2];
    float V = dx*dy*dz; 
    int intersect;
    gflux[tl_ID] = 0;



    if(x_idx < Ngrid || y_idx < Ngrid || z_idx < Ngrid){

        // get voxel surfaces
        x_surfs[0] = x[x_idx]; x_surfs[1] = x[x_idx+1];
        y_surfs[0] = y[y_idx]; y_surfs[1] = y[y_idx+1];
        z_surfs[0] = z[z_idx]; z_surfs[1] = z[z_idx+1];

        for (int PID=0; PID<N; PID++){
            intersect = 1;
            // get particle track length
            tl = track_length[PID];
            // inverted direction to be used in ray-box intersection check
            dir_inv[0] = 1 / u[PID];
            dir_inv[1] = 1 / v[PID];
            dir_inv[2] = 1 / w[PID];
            // default assumption is we cross into box
            x_0 = x_pos[PID]; y_0 = y_pos[PID]; z_0 = z_pos[PID];
            
            // x goes first
            tmin = (x_surfs[0] - x_0) * dir_inv[0];
            tmax = (x_surfs[1] - x_0) * dir_inv[0];
            
            // if necessary swap within x
            if (tmax < tmin){
                savet = tmax;
                tmax = tmin;
                tmin = savet;
            }
            // distance to cross in y
            float tymin = (y_surfs[0] - y_0) * dir_inv[1];
            float tymax = (y_surfs[1] - y_0) * dir_inv[1];
            // if necessary swap within y
            if (tymax < tymin){
                savet = tymax;
                tymax = tymin;
                tymin = savet;
            }

            if ((tmin > tymax) || (tymin > tmax)){
                intersect = 0;
            }

            // swap in between x and y if necesary
            if (tymin > tmin){
                tmin = tymin;
            }
            if (tymax < tmax){
                tmax = tymax;
            }
            
            // distance to cross in z
            float tzmin = (z_surfs[0] - z_0) * dir_inv[2];
            float tzmax = (z_surfs[1] - z_0) * dir_inv[2];
            // if necessary swap within z
            if (tzmax < tzmin){
                savet = tzmax;
                tzmax = tzmin;
                tzmin = savet;
            }

            if ((tmin > tzmax) || (tzmin > tmax)){
                intersect  = 0;
            }

            // swap in between y and z if necessary
            if (tzmin > tmin){
                tmin = tzmin;
            }
            if (tzmax < tmax){
                tmax = tzmax;
            }

            // select cases only where particle was in voxel
            if ( tl > tmin && intersect == 1){
                // particle through entire voxel
                if (tl > tmax && tmin > 0){gflux[tl_ID] += fabsf(tmax - tmin) / V;}
                // particle starts inside voxel, leaves
                if (tmin < 0 && tl > tmax){gflux[tl_ID] += fabsf(tmax) / V;}
                // particle starts outside voxel, end insid
                if (tmax > tl && tmin > 0){gflux[tl_ID] += fabsf(tl - tmin) / V;}
                
                // particle starts inside, ends inside
                if (tmax > tl && tmin < 0){gflux[tl_ID] += tl / V;}

            }
        } 
    }
}


int main(int argc, char* argv[]){

    if (argc != 4){
        std::cout << "Usage: N_particles N h" << std::endl;
        return 1;
    }
    const unsigned Np = atoi(argv[1]);
    const unsigned N = atof(argv[2]);
    const float h = atof(argv[3]); 
    // uniform mesh
    const unsigned NI = N;
    const unsigned NJ = N;
    const unsigned NK = N;
    const float DX = h;
    const float DY = h;
    const float DZ = h;
    // size of flux memory
    int flux_size = N*N*N*sizeof(float);

    if (NI%2 == 0 || NJ%2 == 0 || NK%2 == 0){
        std::cout << "Mesh dimensions must be odd!" << std::endl;
        return 1;
    }
    // generate track histories
    execute_walk(Np);
    // Load particle collision history
    particleTrack hdata = read_array("event_history.txt");
    // generate mesh
    twoDmesh hmesh = gen_mesh(NI, NJ, NK, DX, DY, DZ);
    
    particleTrack ddata = AllocatePtracData(hdata);
    twoDmesh dmesh = AllocateMeshData(hmesh);
    CopyDatatoDevice(ddata, hdata, dmesh, hmesh);

    int max_dim = 10;
    int grid_dim = (int) N*N*N/1000+1; //N/max_dim + 1;
    if (N < 11){
        grid_dim = 1;
        max_dim = N;
    }
    std::cout << "block dim " << max_dim << std::endl;
    std::cout << "N blocks " << grid_dim << std::endl;
    dim3 dimBlock(max_dim, max_dim, max_dim);
    //dim3 dimGrid(grid_dim, grid_dim, grid_dim);
    dim3 dimGrid(grid_dim, 1, 1);
    

    parallel_walk<<<dimGrid,dimBlock>>> (N, ddata.Ntracks, dmesh.x, dmesh.y, dmesh.z, 
            DX, DY, DZ, dmesh.flux, ddata.x_pos, ddata.y_pos, ddata.z_pos,
         ddata.u, ddata.v, ddata.w, ddata.track_length);

    hipMemcpy(hmesh.flux, dmesh.flux, flux_size, 
			hipMemcpyDeviceToHost);
    
    for(int i=0;i<N*N*N;i++){
        std::cout << hmesh.flux[i] << std::endl;
    }

    // hipFree(dmesh.flux);
    // hipFree(dmesh.x);
    // hipFree(dmesh.y);
    // hipFree(dmesh.z);
    // hipFree(ddata.x_pos);
    // hipFree(ddata.y_pos);
    // hipFree(ddata.z_pos);
    // hipFree(ddata.u);
    // hipFree(ddata.v);
    // hipFree(ddata.w);
    // hipFree(ddata.track_length);

    //hipFree(dmesh.flux);
    //hipFree(ddata);
    // free(hmesh.flux);
    return 0;
}
